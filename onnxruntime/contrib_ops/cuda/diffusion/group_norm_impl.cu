#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// The CUDA kernel is modified from GroupNorm plugin of TensorRT 8.5
// Modifications: support more cPerBlock
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/diffusion/group_norm_impl.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

namespace {
constexpr static int32_t CHANNELS_PER_THREAD = 2;  // 2 channels per thread

constexpr static int kSizes[] = {64, 128, 256, 320, 384, 512};
constexpr static size_t kNumOfSizes = sizeof(kSizes) / sizeof(kSizes[0]);
constexpr static int kMaxSize = kSizes[kNumOfSizes - 1];

int nextSize(int x) {
  assert(x <= kMaxSize);

  for (size_t i = 0; i < kNumOfSizes; ++i) {
    if (x <= kSizes[i]) {
      return kSizes[i];
    }
  }

  return kMaxSize;
}
}  // namespace

static inline int32_t divUp(int32_t m, int32_t n) {
  return (m + n - 1) / n;
}

static inline __device__ __host__ float sigmoid(float x) {
  return 1.F / (1.F + expf(-x));
}

struct GroupSums {
  // Is it the 1st element of the group?
  int32_t flag;
  // The sum.
  float sum;
  // The sum of squares.
  float sumSq;
};

struct GroupSumsOp {
  inline __device__ GroupSums operator()(GroupSums const& a, GroupSums const& b) {
    GroupSums dst;
    dst.sum = b.flag ? b.sum : (a.sum + b.sum);
    dst.sumSq = b.flag ? b.sumSq : (a.sumSq + b.sumSq);
    dst.flag = a.flag + b.flag;
    return dst;
  }
};

template <typename T>
struct GroupNormNHWCParams {
  // The output buffer. Shape is (n, h, w, c)
  T* dst;
  // Optional output of element-wise add result of src, skip and bias. Shape is (n, h, w, c) for SkipGroupNorm
  T* add_out;
  // The input buffer. Shape is (n, h, w, c)
  T const* src;
  // Optional input buffer for skip. Shape is (n, h, w, c) for SkipGroupNorm
  T const* skip;
  // Optional input buffer for bias. Shape is (c) for SkipGroupNorm or (n, 1, 1, c) for BiasGroupNorm
  T const* bias;

  // The gamma scaling factor.
  float const* gamma;
  // The beta term to add in GN.
  float const* beta;
  // The temporary buffer to do the global parallel reduction. Size is n x g x 2, where g is number of groups.
  float* redBuffer;

  // The number of instances in the batch.
  int32_t n;
  // The height and width of each activation map.
  int32_t h;
  int32_t w;
  // The number of channels.
  int32_t c;
  // The number of groups.
  int32_t groups;
  // Do we apply the Swish activation function?
  bool withSwish;

  // Precomputed values and parameters to control the execution of the kernels.

  // The number of activations per instance (h * w) and the number of
  // activations per block.
  int32_t hw;
  int32_t hwPerBlock;
  // The number of channels per group and blocks per activation in the C
  // dimension.
  int32_t cPerBlock;
  int32_t cPerGroup;

  // The precomputed stride between instances.
  int32_t hwc;
  // The inverse of hwc in floats (to compute mean/var).
  float invHWC;
  // The precomputed number of groups per block.
  int32_t groupsPerBlock;

  // Number of threads per block
  int32_t threadsPerBlock;

  float epsilon;
};

template <typename T>
inline __device__ void UpdateSum(const T* src, int64_t offset, float& sum, float& sumSq);

template <>
inline __device__ void UpdateSum(const half* src, int64_t offset, float& sum, float& sumSq) {
  // Fetch two channels per thread.
  __half2 h2 = *reinterpret_cast<__half2 const*>(&src[offset]);

  float2 f2 = __half22float2(h2);

  // Update the sum.
  sum += f2.x + f2.y;

  // Update the sum of squares.
  sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <>
inline __device__ void UpdateSum(const float* src, int64_t offset, float& sum, float& sumSq) {
  // Fetch two channels per thread.
  float2 f2 = *reinterpret_cast<float2 const*>(&src[offset]);

  // Update the sum.
  sum += f2.x + f2.y;

  // Update the sum of squares.
  sumSq += f2.x * f2.x + f2.y * f2.y;
}

// Sum for SkipGroupNorm with additional output add_out[offset] = src[offset] + skip[offset] + bias[bias_offset]
template <typename T>
inline __device__ void AddSkipBias(const T* src, const T* skip, const T* bias, T* add_out,
                                   int64_t offset, int32_t bias_offset, float& sum, float& sumSq);

template <>
inline __device__ void AddSkipBias(const half* src, const half* skip, const half* bias, half* add_out,
                                   int64_t offset, int32_t bias_offset, float& sum, float& sumSq) {
  // Fetch two channels per thread.
  __half2 h2 = *reinterpret_cast<__half2 const*>(&src[offset]);
  __half2 s = *reinterpret_cast<__half2 const*>(&skip[offset]);
  __half2 b = *reinterpret_cast<__half2 const*>(&bias[bias_offset]);
  h2 += s;
  h2 += b;

  *reinterpret_cast<__half2*>(&add_out[offset]) = h2;

  float2 f2 = __half22float2(h2);
  sum += f2.x + f2.y;
  sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <>
inline __device__ void AddSkipBias(const float* src, const float* skip, const float* bias, float* add_out,
                                   int64_t offset, int32_t bias_offset, float& sum, float& sumSq) {
  float2 f2 = *reinterpret_cast<float2 const*>(&src[offset]);
  float2 s = *reinterpret_cast<float2 const*>(&skip[offset]);
  float2 b = *reinterpret_cast<float2 const*>(&bias[bias_offset]);
  f2.x += s.x + b.x;
  f2.y += s.y + b.y;

  *reinterpret_cast<float2*>(&add_out[offset]) = f2;

  sum += f2.x + f2.y;
  sumSq += f2.x * f2.x + f2.y * f2.y;
}

// Sum for BiasGroupNorm
template <typename T>
inline __device__ void AddBias(const T* src, const T* bias,
                               int64_t offset, int32_t bias_offset, float& sum, float& sumSq);

template <>
inline __device__ void AddBias(const half* src, const half* bias,
                               int64_t offset, int32_t bias_offset, float& sum, float& sumSq) {
  __half2 h2 = *reinterpret_cast<__half2 const*>(&src[offset]);
  __half2 b = *reinterpret_cast<__half2 const*>(&bias[bias_offset]);
  h2 += b;
  float2 f2 = __half22float2(h2);
  sum += f2.x + f2.y;
  sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <>
inline __device__ void AddBias(const float* src, const float* bias,
                               int64_t offset, int32_t bias_offset, float& sum, float& sumSq) {
  float2 f2 = *reinterpret_cast<float2 const*>(&src[offset]);
  float2 b = *reinterpret_cast<float2 const*>(&bias[bias_offset]);
  f2.x += b.x;
  f2.y += b.y;
  sum += f2.x + f2.y;
  sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <typename T, int32_t THREADS_PER_BLOCK>
__global__ void groupNormNHWCSumKernel(GroupNormNHWCParams<T> params) {
  // The object in charge of doing the sums for the different blocks.
  typedef hipcub::BlockScan<GroupSums, THREADS_PER_BLOCK> BlockScan;

  // Allocate shared memory for BlockScan.
  __shared__ typename BlockScan::TempStorage tempStorage;

  // Allocate shared memory for the groups. We could reduce the amount of shared memory reserved.
  __shared__ float2 smem[THREADS_PER_BLOCK];

  // The instance in the batch.
  int32_t ni = blockIdx.z;

  // The channel loaded by that thread.
  int32_t ci = blockIdx.x * params.cPerBlock + threadIdx.x * CHANNELS_PER_THREAD;

  if (ci >= params.c || threadIdx.x * CHANNELS_PER_THREAD >= params.cPerBlock) {
    return;
  }

  // The first activation loaded by that block.
  int32_t hwBegin = blockIdx.y * params.hwPerBlock;
  // The last activation loaded by that block.
  int32_t hwEnd = min(hwBegin + params.hwPerBlock, params.hw);

  // The sums.
  float sum = 0.F;
  float sumSq = 0.F;

  // Iterate over the activations to compute the sums.
  // We have 3 operators:
  // (1) SkipGroupNorm: skip is (n, h, w, c) and bias is (c), add_out is (n, h, w, c)
  //     The additional output add_out = src + skip + bias.
  // (2) BiasGroupNorm: bias is (n, c), add_out and skip are empty
  // (3) GroupNorm:  skip, bias and add_out not exists

  int64_t offset = static_cast<int64_t>(ni) * params.hwc + static_cast<int64_t>(hwBegin) * params.c + ci;
  if (params.skip != nullptr) {  // SkipGroupNorm
    const int64_t bias_offset = static_cast<int64_t>(ci);
    for (int32_t hwi = hwBegin; hwi < hwEnd; ++hwi, offset += params.c) {
      AddSkipBias(params.src, params.skip, params.bias, params.add_out, offset, bias_offset, sum, sumSq);
    }
  } else if (params.bias != nullptr) {  // BiasGroupNorm
    const int64_t bias_offset = static_cast<int64_t>(ni) * params.c + ci;
    for (int32_t hwi = hwBegin; hwi < hwEnd; ++hwi, offset += params.c) {
      AddBias(params.src, params.bias, offset, bias_offset, sum, sumSq);
    }
  } else {  // GroupNorm
    for (int32_t hwi = hwBegin; hwi < hwEnd; ++hwi, offset += params.c) {
      UpdateSum(params.src, offset, sum, sumSq);
    }
  }

  // The group index relative to the first group within the same block.
  int32_t gi = threadIdx.x * CHANNELS_PER_THREAD / params.cPerGroup;
  // The channel in the group.
  int32_t cj = ci % params.cPerGroup;

  // The data for the summations.
  GroupSums inp{cj == 0 ? 1 : 0, sum, sumSq};

  // Do the segmented scan. InclusiveScan is not deterministic.
  GroupSums out;
  BlockScan(tempStorage).InclusiveScan(inp, out, GroupSumsOp());

  // Store the results for the groups in shared memory (to produce coalesced stores later).
  // For each group, only the last thread of that group is picked to save sum to shared memory and update red buffer.
  const bool is_last_of_a_group = (cj == params.cPerGroup - CHANNELS_PER_THREAD);
  if (is_last_of_a_group) {
    smem[gi] = make_float2(out.sum, out.sumSq);
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  if (is_last_of_a_group) {
    int32_t gj = ci / params.cPerGroup;  // absolute group index
    float2 sums = smem[gi];
    atomicAdd(&params.redBuffer[(2 * ni + 0) * params.groups + gj], sums.x);
    atomicAdd(&params.redBuffer[(2 * ni + 1) * params.groups + gj], sums.y);
  }
}

template <typename T>
void groupNormNHWCSum(GroupNormNHWCParams<T> const& params, hipStream_t stream) {
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = params.c / params.cPerBlock;

  // The number of blocks to compute all the activations in a given instance.
  grid.y = divUp(params.hw, params.hwPerBlock);

  // The number of instances.
  grid.z = params.n;

  // Threads_per_block is half of values in kSizes since CHANNELS_PER_THREAD = 2.
  switch (params.threadsPerBlock) {
    case 256:
      groupNormNHWCSumKernel<T, 256><<<grid, 256, 0, stream>>>(params);
      break;
    case 192:
      groupNormNHWCSumKernel<T, 192><<<grid, 192, 0, stream>>>(params);
      break;
    case 160:
      groupNormNHWCSumKernel<T, 160><<<grid, 160, 0, stream>>>(params);
      break;
    case 128:
      groupNormNHWCSumKernel<T, 128><<<grid, 128, 0, stream>>>(params);
      break;
    case 64:
      groupNormNHWCSumKernel<T, 64><<<grid, 64, 0, stream>>>(params);
      break;
    case 32:
      groupNormNHWCSumKernel<T, 32><<<grid, 32, 0, stream>>>(params);
      break;
  }
}

template <typename T>
__device__ void computeGroupNorm(const T* src, T* dst, int64_t offset, float mean, float invStdDev, float2& gammaF2, float2& betaF2, bool swish);

template <>
__device__ void computeGroupNorm(const half* src, half* dst, int64_t offset, float mean, float invStdDev,
                                 float2& gammaF2, float2& betaF2, bool swish) {
  // Fetch two channels per thread.
  __half2 h2 = *reinterpret_cast<__half2 const*>(&src[offset]);

  // Extract the two half values.
  float2 f2 = __half22float2(h2);

  // Normalize the channels.
  f2.x = (f2.x - mean) * invStdDev;
  f2.y = (f2.y - mean) * invStdDev;

  // Scale by gamma and add beta.
  f2.x = gammaF2.x * f2.x + betaF2.x;
  f2.y = gammaF2.y * f2.y + betaF2.y;

  // Apply SiLU (also known as Swish) if needed.
  if (swish) {
    f2.x = f2.x * sigmoid(f2.x);
    f2.y = f2.y * sigmoid(f2.y);
  }

  *reinterpret_cast<__half2*>(&dst[offset]) = __float22half2_rn(f2);
}

template <>
__device__ void computeGroupNorm(const float* src, float* dst, int64_t offset, float mean, float invStdDev,
                                 float2& gammaF2, float2& betaF2, bool swish) {
  // Fetch two channels per thread.
  float2 f2 = *reinterpret_cast<float2 const*>(&src[offset]);

  // Normalize the channels.
  f2.x = (f2.x - mean) * invStdDev;
  f2.y = (f2.y - mean) * invStdDev;

  // Scale by gamma and add beta.
  f2.x = gammaF2.x * f2.x + betaF2.x;
  f2.y = gammaF2.y * f2.y + betaF2.y;

  // Apply SiLU (also known as Swish) if needed.
  if (swish) {
    f2.x = f2.x * sigmoid(f2.x);
    f2.y = f2.y * sigmoid(f2.y);
  }

  *reinterpret_cast<float2*>(&dst[offset]) = f2;
}

template <typename T>
__global__ void groupNormNHWCScaleKernel(GroupNormNHWCParams<T> params) {
  // The channel loaded by that thread.
  int32_t ci = blockIdx.x * params.cPerBlock + threadIdx.x * CHANNELS_PER_THREAD;
  if (ci >= params.c || threadIdx.x * CHANNELS_PER_THREAD >= params.cPerBlock) {
    return;
  }

  // The instance in the batch.
  int32_t ni = blockIdx.z;

  // The group that thread works on and the channel in the group (modulus).
  int32_t gi = ci / params.cPerGroup;

  // Load the sum and sum of squares for the group.
  float sum = 0.F, sumSq = 0.F;
  if (gi < params.groups) {
    sum = params.redBuffer[(2 * ni + 0) * params.groups + gi];
    sumSq = params.redBuffer[(2 * ni + 1) * params.groups + gi];
  }

  // Load gamma/beta.
  float2 gammaF2 = *reinterpret_cast<float2 const*>(&params.gamma[ci]);
  float2 betaF2 = *reinterpret_cast<float2 const*>(&params.beta[ci]);

  // Compute the mean.
  float mean = sum * params.invHWC;
  // Compute the variance.
  float var = sumSq * params.invHWC - (mean * mean);
  // Compute the inverse of the stddev.
  float invStdDev = rsqrtf(var + params.epsilon);

  // The first activation loaded by that block.
  int32_t hwBegin = blockIdx.y * params.hwPerBlock;
  // The last activation loaded by that block.
  int32_t hwEnd = min(hwBegin + params.hwPerBlock, params.hw);

  // Iterate over the activations to compute the sums.
  for (int32_t hwi = hwBegin; hwi < hwEnd; ++hwi) {
    // The src/dst offset.
    int64_t offset = (int64_t)ni * params.hwc + hwi * params.c + ci;

    // Fetch two channels per thread.
    computeGroupNorm<T>(params.src, params.dst, offset, mean, invStdDev, gammaF2, betaF2, params.withSwish);
  }
}

template <typename T>
void groupNormNHWCScale(GroupNormNHWCParams<T> const& params, hipStream_t stream) {
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = params.c / params.cPerBlock;
  // The number of blocks to compute all the activations in a given instance.
  grid.y = divUp(params.hw, params.hwPerBlock);
  // The number of instances.
  grid.z = params.n;

  switch (params.threadsPerBlock) {
    case 256:
      groupNormNHWCScaleKernel<T><<<grid, 256, 0, stream>>>(params);
      break;
    case 192:
      groupNormNHWCScaleKernel<T><<<grid, 192, 0, stream>>>(params);
      break;
    case 160:
      groupNormNHWCScaleKernel<T><<<grid, 160, 0, stream>>>(params);
      break;
    case 128:
      groupNormNHWCScaleKernel<T><<<grid, 128, 0, stream>>>(params);
      break;
    case 64:
      groupNormNHWCScaleKernel<T><<<grid, 64, 0, stream>>>(params);
      break;
    case 32:
      groupNormNHWCScaleKernel<T><<<grid, 32, 0, stream>>>(params);
      break;
  }
}

int32_t findMaxDivisor(int32_t n, int32_t maxAllowedDivisor) {
  int32_t maxDivisor = -1;
  for (int32_t i = 1; i <= std::sqrt(n); i++) {
    if (n % i == 0) {
      int32_t divisor1 = n / i;
      int32_t divisor2 = i;

      if (divisor1 > maxDivisor && divisor1 < maxAllowedDivisor) {
        maxDivisor = divisor1;
      }
      if (divisor2 > maxDivisor && divisor2 < maxAllowedDivisor) {
        maxDivisor = divisor2;
      }
    }
  }
  return maxDivisor;
}

template <typename T>
Status LaunchGroupNormKernel(
    hipStream_t stream,
    T* output,
    T* add_out,
    const T* input,
    const T* skip,
    const T* bias,
    const float* gamma,
    const float* beta,
    void* workspace,
    float epsilon,
    int batch_size,
    int num_channels,
    int height,
    int width,
    int num_groups,
    bool use_swish_activation) {
  GroupNormNHWCParams<T> params;

  int32_t cPerGroup = num_channels / num_groups;

  int32_t cPerBlock;
  switch (num_channels) {
    case 2560:
    case 2048:
    case 1024:
      cPerBlock = 512;
      break;
    case 1920:
    case 960:
      cPerBlock = 480;
      break;
    case 3072:
    case 1536:
    case 768:
    case 384:
      cPerBlock = 384;
      break;
    case 512:
    case 256:
      cPerBlock = 256;
      break;
    case 2304:
    case 1152:
      cPerBlock = 288;
      break;
    case 128:
      cPerBlock = 128;
      break;
    default:
      cPerBlock = 320;
      if (num_channels % cPerBlock != 0 || cPerBlock % cPerGroup != 0) {
        // Find a maximum cPerBlock that num_channels could be divisible by it.
        // Try to be close to 512 since multiple kSizes values within [256, 512] range could act as fallback.
        cPerBlock = findMaxDivisor(num_groups, kMaxSize / cPerGroup) * cPerGroup;
      }
  }

  params.withSwish = use_swish_activation;
  params.dst = output;
  params.add_out = add_out;
  params.src = input;
  params.skip = skip;
  params.bias = bias;
  params.gamma = gamma;
  params.beta = beta;
  params.redBuffer = reinterpret_cast<float*>(workspace);
  params.n = batch_size;
  params.h = height;
  params.w = width;
  params.c = num_channels;
  params.groups = num_groups;
  params.hw = params.h * params.w;

  // This will allocate as many blocks as possible to partition HW.
  constexpr int32_t maxBlocksPerHW = 1024;
  const int32_t blocksPerHW = findMaxDivisor(params.hw, maxBlocksPerHW);
  params.hwPerBlock = divUp(params.hw, blocksPerHW);

  params.cPerBlock = cPerBlock;
  params.cPerGroup = cPerGroup;
  params.hwc = params.hw * params.c;
  params.invHWC = 1.F / (float)(params.hw * params.cPerGroup);
  params.groupsPerBlock = cPerBlock / params.cPerGroup;
  params.epsilon = epsilon;

  // TODO: Update the kernel to support CHANNELS_PER_THREAD==1 and other corner cases
  if (params.c % params.cPerBlock != 0 ||
      params.cPerBlock % params.cPerGroup != 0 ||
      cPerBlock > 512 ||
      (params.cPerGroup % CHANNELS_PER_THREAD != 0)) {
    printf("n=%d h=%d w=%d c=%d groups=%d hw=%d hwPerBlock=%d cPerBlock=%d cPerGroup=%d\n",
           params.n, params.h, params.w, params.c, params.groups, params.hw, params.hwPerBlock,
           params.cPerBlock, params.cPerGroup);
    ORT_NOT_IMPLEMENTED("Not implemented");
  }

  params.threadsPerBlock = nextSize(cPerBlock) / CHANNELS_PER_THREAD;

#ifdef DUMP_GROUP_NORM
  printf("n=%d h=%d w=%d c=%d groups=%d hw=%d hwPerBlock=%d cPerBlock=%d cPerGroup=%d threadsPerBlock=%d\n",
         params.n, params.h, params.w, params.c, params.groups, params.hw, params.hwPerBlock,
         params.cPerBlock, params.cPerGroup, params.threadsPerBlock);
#endif

  CUDA_RETURN_IF_ERROR(hipMemsetAsync(params.redBuffer, 0, GetGroupNormWorkspaceSizeInBytes(batch_size, num_groups), stream));

  groupNormNHWCSum<T>(params, stream);
  CUDA_RETURN_IF_ERROR(hipGetLastError());

  groupNormNHWCScale<T>(params, stream);
  CUDA_RETURN_IF_ERROR(hipGetLastError());

  return Status::OK();
}

template Status LaunchGroupNormKernel<half>(hipStream_t stream, half* output, half* add_out,
                                            const half* input, const half* skip, const half* bias,
                                            const float* gamma, const float* beta, void* workspace,
                                            float epsilon, int batch_size, int num_channels,
                                            int height, int width, int num_groups, bool swish);

template Status LaunchGroupNormKernel<float>(hipStream_t stream, float* output, float* add_out,
                                             const float* input, const float* skip, const float* bias,
                                             const float* gamma, const float* beta, void* workspace,
                                             float epsilon, int batch_size, int num_channels,
                                             int height, int width, int num_groups, bool swish);
}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
